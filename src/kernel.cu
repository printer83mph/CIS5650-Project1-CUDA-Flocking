#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA

#include "kernel.cuh"
#include <hip/hip_runtime.h>

#include <cstdio>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/sort.h>

#include <glm/glm.hpp>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax(a, b) (((a) > (b)) ? (a) : (b))
#endif

#ifndef imin
#define imin(a, b) (((a) < (b)) ? (a) : (b))
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
 * Check for CUDA errors; print and exit if there was a problem.
 */
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/*****************
 * Configuration *
 *****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
 * Kernel state (pointers are device pointers) *
 ***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents
                               // this particle?
int *dev_particleGridIndices;  // What grid cell is this particle in?

// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// Used for sorting array data for faster access
glm::vec3 *dev_pos_sorted;
glm::vec3 *dev_vel1_sorted;

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
 * initSimulation *
 ******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
 * LOOK-1.2 - this is a typical helper function for a CUDA kernel.
 * Function for generating a random vec3.
 */
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng),
                   (float)unitDistrib(rng));
}

/**
 * LOOK-1.2 - This is a basic CUDA kernel.
 * CUDA kernel for generating boids with a specified mass randomly around the
 * star.
 */
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 *arr,
                                           float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
 * Initialize memory, update some globals
 */
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void **)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void **)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void **)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(
      1, numObjects, dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth =
      2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  hipMalloc((void **)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void **)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  dev_thrust_particleArrayIndices =
      thrust::device_pointer_cast(dev_particleArrayIndices);
  dev_thrust_particleGridIndices =
      thrust::device_pointer_cast(dev_particleGridIndices);

  hipMalloc((void **)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void **)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void **)&dev_pos_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_sorted failed!");

  hipMalloc((void **)&dev_vel1_sorted, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1_sorted failed!");

  hipDeviceSynchronize();
}

/******************
 * copyBoidsToVBO *
 ******************/

/**
 * Copy the boid positions into the VBO so that they can be drawn by OpenGL.
 */
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo,
                                       float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo,
                                        float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
 * Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
 */
void Boids::copyBoidsToVBO(float *vbodptr_positions,
                           float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}

/******************
 * stepSimulation *
 ******************/

/**
 * Quick utility for getting the square of a number at compile time.
 * Used for magnitude calculations.
 */
__device__ constexpr float square(float n) { return n * n; }

/**
 * LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
 * __device__ code can be called from a __global__ context
 * Compute the new velocity on the body with index `iSelf` due to the `N` boids
 * in the `pos` and `vel` arrays.
 */
__device__ glm::vec3 computeVelocityChange(int N, int iSelf,
                                           const glm::vec3 *pos,
                                           const glm::vec3 *vel) {
  glm::vec3 posSelf = pos[iSelf];
  glm::vec3 totalVelocityChange = glm::vec3(0.0f, 0.0f, 0.0f);

  // Rule 1: boids fly towards their local perceived center of mass, which
  // excludes themselves
  glm::vec3 perceivedCenterOfMass = glm::vec3(0.0f, 0.0f, 0.0f);
  int massNeighbors = 0;
  for (int i = 0; i < N; ++i) {
    glm::vec3 posI = pos[i];
    glm::vec3 distance = posI - posSelf;
    if (i == iSelf || (glm::dot(distance, distance) > square(rule1Distance)))
      continue;

    massNeighbors++;
    perceivedCenterOfMass += posI;
  }
  if (massNeighbors > 0) {
    perceivedCenterOfMass /= massNeighbors;
    totalVelocityChange += (perceivedCenterOfMass - posSelf) * rule1Scale;
  }

  // Rule 2: boids try to stay a distance d away from each other
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);

  for (int i = 0; i < N; ++i) {
    glm::vec3 posI = pos[i];
    glm::vec3 distance = posI - posSelf;
    if (i == iSelf || (glm::dot(distance, distance) > square(rule2Distance)))
      continue;

    c -= distance;
  }
  totalVelocityChange += c * rule2Scale;

  // Rule 3: boids try to match the speed of surrounding boids
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
  int velocityNeighbors = 0;
  for (int i = 0; i < N; ++i) {
    glm::vec3 posI = pos[i];
    glm::vec3 distance = posI - posSelf;
    if (i == iSelf || (glm::dot(distance, distance) > square(rule3Distance)))
      continue;

    velocityNeighbors++;
    perceivedVelocity += vel[i];
  }
  if (velocityNeighbors > 0) {
    perceivedVelocity /= velocityNeighbors;
    totalVelocityChange += perceivedVelocity * rule3Scale;
  }

  // Return total velocity change
  return totalVelocityChange;
}

/**
 * TODO-1.2 implement basic flocking
 * For each of the `N` bodies, update its position based on its current
 * velocity.
 */
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
                                             glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisVel = vel1[index];
  glm::vec3 newVel = thisVel + computeVelocityChange(N, index, pos, vel1);

  // Clamp the speed
  float newSpeedSquared = glm::dot(newVel, newVel);
  if (newSpeedSquared > square(maxSpeed)) {
    newVel = newVel / sqrt(newSpeedSquared) * maxSpeed;
  }

  // Record the new velocity into vel2. Question: why NOT vel1?
  vel2[index] = newVel;
}

/**
 * LOOK-1.2 Since this is pretty trivial, we implemented it for you.
 * For each of the `N` bodies, update its position based on its current
 * velocity.
 */
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
//
// Note to self - match largest factors with outermost for loops, such that
// the iteration goes through indices contiguously
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__device__ glm::ivec3 gridIndex1Dto3D(int index, int gridResolution) {
  return glm::ivec3(index % gridResolution,
                    (index / gridResolution) % gridResolution,
                    (index / (gridResolution * gridResolution)));
}

__global__ void kernComputeIndices(int N, int gridResolution, glm::vec3 gridMin,
                                   float inverseCellWidth, glm::vec3 *pos,
                                   int *indices, int *gridIndices) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N)
    return;

  glm::vec3 posSelf = pos[index];

  glm::vec3 gridPos = glm::floor((posSelf - gridMin) * inverseCellWidth);

  indices[index] = index;
  gridIndices[index] =
      gridIndex3Dto1D(gridPos.x, gridPos.y, gridPos.z, gridResolution);
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
                                         int *gridCellStartIndices,
                                         int *gridCellEndIndices) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N)
    return;

  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"

  int selfGridIndex = particleGridIndices[index];

  if (index == 0)
    gridCellStartIndices[selfGridIndex] = 0;

  else if (index == N - 1)
    gridCellEndIndices[selfGridIndex] = N;

  else {
    int prevGridIndex = particleGridIndices[index - 1];
    if (prevGridIndex != selfGridIndex) {
      gridCellEndIndices[prevGridIndex] = index;
      gridCellStartIndices[selfGridIndex] = index;
    }
  }
}

__device__ constexpr float cxpr_max(float a, float b) { return a > b ? a : b; }

// Update a boid's velocity using the uniform grid to reduce
// the number of boids that need to be checked.
__global__ void kernUpdateVelNeighborSearchScattered(
    int N, int gridResolution, glm::vec3 gridMin, float inverseCellWidth,
    float cellWidth, int *gridCellStartIndices, int *gridCellEndIndices,
    int *particleArrayIndices, glm::vec3 *pos, glm::vec3 *vel1,
    glm::vec3 *vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // This gets added to by all goobers
  glm::vec3 totalAddedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

  glm::vec3 posSelf = pos[index];
  glm::vec3 velSelf = vel1[index];

  // Identify which cells may contain neighbors. This isn't always 8. Let's set
  // the bounds based on our particle's position and the neighbor search radius.

  glm::ivec3 cellPosSelf = glm::floor((posSelf - gridMin) * inverseCellWidth);
  float neighborMaxRadius =
      cxpr_max(cxpr_max(rule1Distance, rule2Distance), rule3Distance);
  glm::vec3 searchLength = glm::vec3(neighborMaxRadius);

  glm::ivec3 gridSearchMin =
      glm::floor((posSelf - gridMin - searchLength) * inverseCellWidth);
  glm::ivec3 gridSearchMaxInclusive =
      glm::floor((posSelf - gridMin + searchLength) * inverseCellWidth);

  // Rule 1 data collection
  glm::vec3 perceivedCenterOfMass = glm::vec3(0.0f, 0.0f, 0.0f);
  int centerOfMassNeighbors = 0;
  // Rule 2 data collection
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);
  // RUle 3 data collection
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
  int velocityNeighbors = 0;

  // Iterate through all possibly influential cells
  for (int z = gridSearchMin.z; z <= gridSearchMaxInclusive.z; z++) {
    for (int y = gridSearchMin.y; y <= gridSearchMaxInclusive.y; y++) {
      for (int x = gridSearchMin.x; x <= gridSearchMaxInclusive.x; x++) {
        glm::ivec3 neighborCellPos = glm::ivec3(x, y, z);

        // Skip iteration if outside bounds
        if (neighborCellPos.x < 0 || neighborCellPos.x >= gridResolution ||
            neighborCellPos.y < 0 || neighborCellPos.y >= gridResolution ||
            neighborCellPos.z < 0 || neighborCellPos.z >= gridResolution) {
          continue;
        }

        int neighborGridCell =
            gridIndex3Dto1D(neighborCellPos.x, neighborCellPos.y,
                            neighborCellPos.z, gridResolution);

        // Get boid start/end indices for this cell
        int startIdx = gridCellStartIndices[neighborGridCell];
        int endIdx = gridCellEndIndices[neighborGridCell];

        for (int i = startIdx; i < endIdx; ++i) {
          int bufferIndex = particleArrayIndices[i];
          if (bufferIndex == index)
            continue;

          glm::vec3 posI = pos[bufferIndex];
          glm::vec3 distance = posI - posSelf;
          float distanceSq = glm::dot(distance, distance);

          // Rule 1: boids fly towards their local perceived center of mass,
          // which excludes themselves
          if (distanceSq < square(rule1Distance)) {
            centerOfMassNeighbors++;
            perceivedCenterOfMass += posI;
          }
          // Rule 2: boids try to stay a distance d away from each other
          if (distanceSq < square(rule2Distance)) {
            c -= distance;
          }
          // Rule 3: boids try to match the speed of surrounding boids
          if (distanceSq < square(rule3Distance)) {
            velocityNeighbors++;
            perceivedVelocity += vel1[bufferIndex];
          }
        }
      }
    }
  }

  // Apply rule 1 to overall velocity addition
  if (centerOfMassNeighbors > 0) {
    perceivedCenterOfMass /= centerOfMassNeighbors;
    totalAddedVelocity += (perceivedCenterOfMass - posSelf) * rule1Scale;
  }

  // Apply rule 2 to overall velocity addition
  totalAddedVelocity += c * rule2Scale;

  // Apply rule 3 to overall velocity addition
  if (velocityNeighbors > 0) {
    perceivedVelocity /= velocityNeighbors;
    totalAddedVelocity += perceivedVelocity * rule3Scale;
  }

  glm::vec3 velNew = velSelf + totalAddedVelocity;

  // Clamp the speed
  float newSpeedSquared = glm::dot(velNew, velNew);
  if (newSpeedSquared > square(maxSpeed)) {
    velNew = velNew / sqrt(newSpeedSquared) * maxSpeed;
  }

  // Record the new velocity into vel2
  vel2[index] = velNew;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
    int N, int gridResolution, glm::vec3 gridMin, float inverseCellWidth,
    float cellWidth, int *gridCellStartIndices, int *gridCellEndIndices,
    glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  // This gets added to by all goobers
  glm::vec3 totalAddedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);

  glm::vec3 posSelf = pos[index];
  glm::vec3 velSelf = vel1[index];

  // Identify which cells may contain neighbors. This isn't always 8. Let's set
  // the bounds based on our particle's position and the neighbor search radius.

  glm::ivec3 cellPosSelf = glm::floor((posSelf - gridMin) * inverseCellWidth);
  float neighborMaxRadius =
      cxpr_max(cxpr_max(rule1Distance, rule2Distance), rule3Distance);
  glm::vec3 searchLength = glm::vec3(neighborMaxRadius);

  glm::ivec3 gridSearchMin =
      glm::floor((posSelf - gridMin - searchLength) * inverseCellWidth);
  glm::ivec3 gridSearchMaxInclusive =
      glm::floor((posSelf - gridMin + searchLength) * inverseCellWidth);

  // Rule 1 data collection
  glm::vec3 perceivedCenterOfMass = glm::vec3(0.0f, 0.0f, 0.0f);
  int centerOfMassNeighbors = 0;
  // Rule 2 data collection
  glm::vec3 c = glm::vec3(0.0f, 0.0f, 0.0f);
  // Rule 3 data collection
  glm::vec3 perceivedVelocity = glm::vec3(0.0f, 0.0f, 0.0f);
  int velocityNeighbors = 0;

  // Iterate through all possibly influential cells
  // For memory efficiency, we go z->y->x (contiguous memory since our indexing
  // is z-major then y-major inside that)
  for (int z = gridSearchMin.z; z <= gridSearchMaxInclusive.z; z++) {
    for (int y = gridSearchMin.y; y <= gridSearchMaxInclusive.y; y++) {
      for (int x = gridSearchMin.x; x <= gridSearchMaxInclusive.x; x++) {
        glm::ivec3 neighborCellPos = glm::ivec3(x, y, z);

        // Skip iteration if outside bounds
        if (neighborCellPos.x < 0 || neighborCellPos.x >= gridResolution ||
            neighborCellPos.y < 0 || neighborCellPos.y >= gridResolution ||
            neighborCellPos.z < 0 || neighborCellPos.z >= gridResolution) {
          continue;
        }

        int neighborGridCell =
            gridIndex3Dto1D(neighborCellPos.x, neighborCellPos.y,
                            neighborCellPos.z, gridResolution);

        // Get boid start/end indices for this cell
        int startIdx = gridCellStartIndices[neighborGridCell];
        int endIdx = gridCellEndIndices[neighborGridCell];

        // Access boids directly (no stupid bufferIndex indirection)
        for (int i = startIdx; i < endIdx; ++i) {
          if (i == index)
            continue;

          glm::vec3 posI = pos[i];
          glm::vec3 distance = posI - posSelf;
          float distanceSq = glm::dot(distance, distance);

          // Rule 1: boids fly towards their local perceived center of mass,
          // which excludes themselves
          if (distanceSq < square(rule1Distance)) {
            centerOfMassNeighbors++;
            perceivedCenterOfMass += posI;
          }
          // Rule 2: boids try to stay a distance d away from each other
          if (distanceSq < square(rule2Distance)) {
            c -= distance;
          }
          // Rule 3: boids try to match the speed of surrounding boids
          if (distanceSq < square(rule3Distance)) {
            velocityNeighbors++;
            perceivedVelocity += vel1[i];
          }
        }
      }
    }
  }

  // Apply rule 1 to overall velocity addition
  if (centerOfMassNeighbors > 0) {
    perceivedCenterOfMass /= centerOfMassNeighbors;
    totalAddedVelocity += (perceivedCenterOfMass - posSelf) * rule1Scale;
  }

  // Apply rule 2 to overall velocity addition
  totalAddedVelocity += c * rule2Scale;

  // Apply rule 3 to overall velocity addition
  if (velocityNeighbors > 0) {
    perceivedVelocity /= velocityNeighbors;
    totalAddedVelocity += perceivedVelocity * rule3Scale;
  }

  glm::vec3 velNew = velSelf + totalAddedVelocity;

  // Clamp the speed
  float newSpeedSquared = glm::dot(velNew, velNew);
  if (newSpeedSquared > square(maxSpeed)) {
    velNew = velNew / sqrt(newSpeedSquared) * maxSpeed;
  }

  // Record the new velocity into vel2
  vel2[index] = velNew;
}

__global__ void kernReorderDataByIndices(int N, int *particleArrayIndices,
                                         glm::vec3 *pos, glm::vec3 *vel1,
                                         glm::vec3 *pos_sorted,
                                         glm::vec3 *vel1_sorted) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index >= N) {
    return;
  }

  int originalIndex = particleArrayIndices[index];
  pos_sorted[index] = pos[originalIndex];
  vel1_sorted[index] = vel1[originalIndex];
}

/**
 * Step the entire N-body simulation by `dt` seconds.
 */
void Boids::stepSimulationNaive(float dt) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelocityBruteForce failed!");

  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos,
                                                  dev_vel2);

  // Ping-pong velocity arrays
  glm::vec3 *originalVel1 = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = originalVel1;
}

void Boids::stepSimulationScatteredGrid(float dt) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // Label each particle with its array index and grid index
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
      dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");

  // Arcane thrust magic to sort arrays using grid indices
  thrust::sort_by_key(dev_thrust_particleGridIndices,
                      dev_thrust_particleGridIndices + numObjects,
                      dev_thrust_particleArrayIndices);

  // Reset grid cell start/end indices
  dim3 gridBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<gridBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellStartIndices, 0);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");
  kernResetIntBuffer<<<gridBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellEndIndices, 0);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");

  // Find start and end indices for each grid cell
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_particleGridIndices, dev_gridCellStartIndices,
      dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

  // Update velocities using scattered grid neighbor search
  kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
      dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchScattered failed!");

  // Update positions
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos,
                                                  dev_vel2);
  checkCUDAErrorWithLine("kernUpdatePos failed!");

  // Ping-pong velocity arrays
  glm::vec3 *originalVel1 = dev_vel1;
  dev_vel1 = dev_vel2;
  dev_vel2 = originalVel1;
}

void Boids::stepSimulationCoherentGrid(float dt) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  // Label each particle with array index and grid index
  kernComputeIndices<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos,
      dev_particleArrayIndices, dev_particleGridIndices);
  checkCUDAErrorWithLine("kernComputeIndices failed!");

  // Run arcane thrust magic
  thrust::sort_by_key(dev_thrust_particleGridIndices,
                      dev_thrust_particleGridIndices + numObjects,
                      dev_thrust_particleArrayIndices);

  // Reshuffle position and velocity data to match with grid cells
  hipMalloc((void **)&dev_pos_sorted, numObjects * sizeof(glm::vec3));
  hipMalloc((void **)&dev_vel1_sorted, numObjects * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc temp buffers failed!");

  kernReorderDataByIndices<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_particleArrayIndices, dev_pos, dev_vel1, dev_pos_sorted,
      dev_vel1_sorted);
  checkCUDAErrorWithLine("kernReorderDataByIndices failed!");

  // Reset grid cell start/end indices
  dim3 gridBlocksPerGrid((gridCellCount + blockSize - 1) / blockSize);
  kernResetIntBuffer<<<gridBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellStartIndices, 0);
  kernResetIntBuffer<<<gridBlocksPerGrid, blockSize>>>(
      gridCellCount, dev_gridCellEndIndices, 0);
  checkCUDAErrorWithLine("kernResetIntBuffer failed!");

  // Find start and end indices for each grid cell
  kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, dev_particleGridIndices, dev_gridCellStartIndices,
      dev_gridCellEndIndices);
  checkCUDAErrorWithLine("kernIdentifyCellStartEnd failed!");

  // Update velocities using coherent grid neighbor search
  kernUpdateVelNeighborSearchCoherent<<<fullBlocksPerGrid, blockSize>>>(
      numObjects, gridSideCount, gridMinimum, gridInverseCellWidth,
      gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices,
      dev_pos_sorted, dev_vel1_sorted, dev_vel2);
  checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherent failed!");

  // Update positions using sorted position data
  kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt,
                                                  dev_pos_sorted, dev_vel2);
  checkCUDAErrorWithLine("kernUpdatePos failed!");

  // Copy sorted data back to original buffers
  hipMemcpy(dev_pos, dev_pos_sorted, numObjects * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
  hipMemcpy(dev_vel1, dev_vel2, numObjects * sizeof(glm::vec3),
             hipMemcpyDeviceToDevice);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]> intKeys{new int[N]};
  std::unique_ptr<int[]> intValues{new int[N]};

  intKeys[0] = 0;
  intValues[0] = 0;
  intKeys[1] = 1;
  intValues[1] = 1;
  intKeys[2] = 0;
  intValues[2] = 2;
  intKeys[3] = 3;
  intValues[3] = 3;
  intKeys[4] = 0;
  intValues[4] = 4;
  intKeys[5] = 2;
  intValues[5] = 5;
  intKeys[6] = 2;
  intValues[6] = 6;
  intKeys[7] = 0;
  intValues[7] = 7;
  intKeys[8] = 5;
  intValues[8] = 8;
  intKeys[9] = 6;
  intValues[9] = 9;

  hipMalloc((void **)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void **)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N,
             hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N,
             hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N,
             hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N,
             hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
